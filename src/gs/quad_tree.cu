#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: 2024 Smart Robotics Lab, Technical University of Munich
 * SPDX-FileCopyrightText: 2024 Jiaxin Wei
 * SPDX-License-Identifier: BSD-3-Clause
 */

#include <numeric>

#include "gs/quad_tree.cuh"

namespace gs {

cv::Mat Node::getPixels(const cv::Mat& img) const
{
    cv::Rect roi(x0_, y0_, width_, height_);
    return img(roi);
}

float Node::computeError(const cv::Mat& img) const
{
    cv::Mat pixels = getPixels(img);
    cv::Scalar avg_color = cv::mean(pixels);

    std::vector<cv::Mat> channels;
    cv::split(pixels, channels);

    float r_mse = 0.0f, g_mse = 0.0f, b_mse = 0.0f;

    for (int i = 0; i < pixels.rows; i++) {
        for (int j = 0; j < pixels.cols; j++) {
            float r_diff = static_cast<float>(channels[0].at<uchar>(i, j)) - avg_color[0];
            float g_diff = static_cast<float>(channels[1].at<uchar>(i, j)) - avg_color[1];
            float b_diff = static_cast<float>(channels[2].at<uchar>(i, j)) - avg_color[2];

            r_mse += r_diff * r_diff;
            g_mse += g_diff * g_diff;
            b_mse += b_diff * b_diff;
        }
    }

    int count = pixels.rows * pixels.cols;
    r_mse /= count;
    g_mse /= count;
    b_mse /= count;

    float error = r_mse * 0.2989 + g_mse * 0.5870 + b_mse * 0.1140;

    return error * img.rows * img.cols / 90000000.0;
}

void QTree::subdivide()
{
    recursive_subdivide(root_, threshold_, min_pixel_size_, img_);
    all_children_ = find_children(root_);
}


void QTree::renderImg(int thickness, cv::Scalar color)
{
    cv::Mat imgc;
    cv::cvtColor(img_, imgc, cv::COLOR_RGB2BGR);
    cv::imshow("before", imgc);

    std::vector<Node> children = find_children(root_);
    std::cout << "Find " << children.size() << " nodes" << std::endl;

    for (const auto& child : children) {
        cv::Mat pixels = child.getPixels(img_);

        cv::Scalar avg_color = cv::mean(pixels);
        int avg_b = static_cast<int>(std::floor(avg_color[0]));
        int avg_g = static_cast<int>(std::floor(avg_color[1]));
        int avg_r = static_cast<int>(std::floor(avg_color[2]));

        imgc(cv::Rect(child.getOriginX(), child.getOriginY(), child.getWidth(), child.getHeight())).setTo(cv::Scalar(avg_r, avg_g, avg_b));
        if (thickness > 0) {
            cv::rectangle(imgc, cv::Point(child.getOriginX(), child.getOriginY()), cv::Point(child.getOriginX() + child.getWidth(), child.getOriginY() + child.getHeight()), color, thickness);
        }
    }

    cv::imshow("after", imgc);
    cv::waitKey(0);
}

void recursive_subdivide(Node& node, float threshold, int min_pixel_size, cv::Mat& img)
{
    if (node.computeError(img) <= threshold) {
        return;
    }

    int w1 = static_cast<int>(std::floor(node.getWidth() / 2.0));
    int w2 = static_cast<int>(std::ceil(node.getWidth() / 2.0));
    int h1 = static_cast<int>(std::floor(node.getHeight() / 2.0));
    int h2 = static_cast<int>(std::ceil(node.getHeight() / 2.0));

    if (w1 <= min_pixel_size || h1 <= min_pixel_size) {
        return;
    }

    // top left
    Node n1(node.getOriginX(), node.getOriginY(), w1, h1);
    recursive_subdivide(n1, threshold, min_pixel_size, img);
    // bottom left
    Node n2(node.getOriginX(), node.getOriginY() + h1, w1, h2);
    recursive_subdivide(n2, threshold, min_pixel_size, img);
    // top right
    Node n3(node.getOriginX() + w1, node.getOriginY(), w2, h1);
    recursive_subdivide(n3, threshold, min_pixel_size, img);
    // bottom right
    Node n4(node.getOriginX() + w1, node.getOriginY() + h1, w2, h2);
    recursive_subdivide(n4, threshold, min_pixel_size, img);

    std::vector<Node> children{n1, n2, n3, n4};
    node.children = children;
}

std::vector<Node> find_children(const Node& node)
{
    if (node.children.empty()) {
        return {node};
    }
    else {
        std::vector<Node> all_children;
        for (const auto& child : node.children) {
            auto grandchildren = find_children(child);
            all_children.insert(all_children.end(), grandchildren.begin(), grandchildren.end());
        }
        return all_children;
    }
}

} // namespace gs